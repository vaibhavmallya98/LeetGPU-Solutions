#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    output[i] = input[i] <= 0 ? 0.01 * input[i] : input[i]; 

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
