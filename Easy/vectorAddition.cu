#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x; 

    if(idx < N){
        C[idx] = A[idx] + B[idx]; 
    }

}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}
