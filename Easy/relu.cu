#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x; 

    if(idx < N){
        output[idx] = max(0.0f,input[idx]);
    }

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
